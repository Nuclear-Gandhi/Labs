#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void CUDACount(char *A, char *B, int *len, int *wordLen, int *cnt)
{
    int idx = threadIdx.x, flag = 1;

    if (idx + *wordLen <= *len)
    {
        for (int i = 0; i < *wordLen; i++)
        {
            if (A[idx + i] != B[i])
            {
                flag = 0;
                break;
            }
        }

        if (flag == 1)
            atomicAdd(cnt, 1);
    }
}

int main()
{
    char A[N], B[N];
    char *d_A, *d_B;

    int count = 0, len, wordLen, res;
    int *d_count, *d_len, *d_wordLen;

    printf("Enter String : ");
    scanf("%[^\n]%*c", A);
    printf("String : %s\n\n", A);

    printf("Enter Word to be searched in String : ");
    scanf("%s", B);
    printf("Word : %s\n\n", B);

    len = strlen(A);
    wordLen = strlen(B);

    hipMalloc((void **)&d_A, strlen(A) * sizeof(char));
    hipMalloc((void **)&d_B, strlen(B) * sizeof(char));
    hipMalloc((void **)&d_count, sizeof(int));
    hipMalloc((void **)&d_len, sizeof(int));
    hipMalloc((void **)&d_wordLen, sizeof(int));
    hipMalloc((void **)&res, sizeof(int));

    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLen, &wordLen, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, strlen(B) * sizeof(char), hipMemcpyHostToDevice);

    CUDACount<<<1, (strlen(A) - strlen(B) + 1)>>>(d_A, d_B, d_len, d_wordLen, d_count);
    hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Occurances of '%s' = %d\n", B, res);
}